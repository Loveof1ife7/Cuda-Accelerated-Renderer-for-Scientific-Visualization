#include "hip/hip_runtime.h"
#include "scene.hpp"
#include "cuda_utils.hpp"
#include <Eigen/Dense>

Scene::~Scene()
{
    if (m_ds_dev)
    {
        CUDA_CHECK(hipFree(m_ds_dev));
    }
}
Scene &Scene::setCamera(const Camera *cam)
{
    m_cam = cam;
    m_dirtyCam = true;
    return *this;
}
Scene &Scene::setVolume(std::shared_ptr<Volume> vol)
{
    m_volume = std::move(vol);
    m_dirtyVol = true;
    return *this;
}
Scene &Scene::setTransferFunction(std::shared_ptr<TransferFunction> tf)
{
    m_tf = std::move(tf);
    m_dirtyTF = true;
    return *this;
}
Scene &Scene::setLights(const Lights *lights)
{
    m_lights = lights;
    m_dirtyLights = true;
    return *this;
}
Scene &Scene::setRenderParams(float step, float opacityScale, int mode, float iso)
{
    m_stepSize = step;
    m_opacityScale = opacityScale;
    m_mode = mode;
    m_isoValue = iso;
    m_dirtyParams = true;
    return *this;
}
Scene &Scene::setClipBox(float3 clip_min, float3 clip_max)
{
    m_clipMin = clip_min;
    m_clipMax = clip_max;
    m_dirtyParams = true;
    return *this;
}
void Scene::validateOrThrow() const
{
    if (!m_cam)
        throw std::runtime_error("Scene: No camera set");
    if (!m_volume)
        throw std::runtime_error("Scene: No volume set");
    if (!m_tf)
        throw std::runtime_error("Scene: No transfer function set");

    if (m_stepSize <= 0.f)
        throw std::runtime_error("Scene: Invalid step size");

    if (m_volume->getDesc().dim.x <= 0 || m_volume->getDesc().dim.y <= 0 || m_volume->getDesc().dim.z <= 0)
        throw std::runtime_error("Scene: invalid volume dimensions.");
}
void Scene::commit(hipStream_t stream)
{
    validateOrThrow();

    if (m_dirtyCam)
    {
        DeviceCamera dc{};
        dc.position_ = camera_utils::f3(m_cam->getPosition());
        dc.forward_ = camera_utils::f3(m_cam->getForward().normalized());
        dc.up_ = camera_utils::f3(m_cam->getUp().normalized());
        dc.right_ = camera_utils::f3(m_cam->getRight().normalized());
        dc.vertical_fov_ = m_cam->getVerticalFov();

        m_ds_host.d_camera = dc;
        m_dirtyCam = false;
    }
    if (m_dirtyVol)
    {
        const auto &d = m_volume->getDesc();
        DeviceVolume dv{};
        dv.field_tex = m_volume->getFieldTex();
        dv.grad_tex = m_volume->getGradTex();
        dv.dim = d.dim;
        dv.voxel_size = d.voxelSize;
        dv.origin = d.origin;
        dv.value_range = d.valueRange;
        dv.density_scale = d.densityScale;

        m_ds_host.d_volume = dv;
        m_dirtyVol = false;
    }
    if (m_dirtyTF)
    {
        DeviceTF d_tf{};
        d_tf.tf1D = m_tf->getCudaTex();
        d_tf.domain = m_tf->getDomain();

        m_ds_host.d_tf = d_tf;
        m_dirtyTF = false;
    }

    if (m_dirtyLights)
    {
        m_ds_host.d_lights = m_lights ? m_lights->getDevicePointer() : nullptr;
        m_ds_host.lights_count = m_lights ? m_lights->count() : 0;
        m_dirtyLights = false;
    }

    // Params
    if (m_dirtyParams)
    {
        m_ds_host.step_size = m_stepSize;
        m_ds_host.opacityScale = m_opacityScale;
        m_ds_host.mode = m_mode;
        m_ds_host.isoValue = m_isoValue;
        m_ds_host.clipMin = m_clipMin;
        m_ds_host.clipMax = m_clipMax;
        m_dirtyParams = false;
    }

    if (!m_ds_dev)
    {
        CUDA_CHECK(
            hipMalloc(&m_ds_dev, sizeof(m_ds_host)));

        CUDA_CHECK(
            hipMemcpyAsync(m_ds_dev, &m_ds_host, sizeof(m_ds_host), hipMemcpyHostToDevice));
    }
}

SceneDebugInfo Scene::debug() const
{
    SceneDebugInfo info{};
    if (!m_volume || !m_tf)
        return info;
    info.valid = true;
    info.dim = m_volume->getDesc().dim;
    info.voxelSize = m_volume->getDesc().voxelSize;
    info.valueRange = m_volume->getDesc().valueRange;
    info.tfSize = 0; // 若需要可在 TF 中保存 N
    info.lightCount = m_lights ? m_lights->count() : 0;
    info.stepSize = m_stepSize;
    info.mode = m_mode;
    return info;
}