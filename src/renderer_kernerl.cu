#include "hip/hip_runtime.h"
#include "renderer_kernel.hpp"

using kernel_utils::add3;
using kernel_utils::clampf;
using kernel_utils::cross3;
using kernel_utils::dot3;
using kernel_utils::f3;
using kernel_utils::f4;
using kernel_utils::length3;
using kernel_utils::mad3;
using kernel_utils::mulS;
using kernel_utils::mulV;
using kernel_utils::normalize3;
using kernel_utils::sub3;

__device__ bool intersectAABB(
    const float3 &ro, const float3 &rd,
    const float3 &bmin, const float3 &bmax,
    float &tmin, float &tmax)
{
    float3 inv;
    inv.x = (fabsf(rd.x) > 1e-8f) ? 1.0f / rd.x : 1e32f;
    inv.y = (fabsf(rd.y) > 1e-8f) ? 1.0f / rd.y : 1e32f;
    inv.z = (fabsf(rd.z) > 1e-8f) ? 1.0f / rd.z : 1e32f;

    const float3 t0 = mulV(sub3(bmin, ro), inv);
    const float3 t1 = mulV(sub3(bmax, ro), inv);

    // 每轴取小/大的那个（进入/离开）
    const float3 tsm = f3(fminf(t0.x, t1.x), fminf(t0.y, t1.y), fminf(t0.z, t1.z));
    const float3 tbg = f3(fmaxf(t0.x, t1.x), fmaxf(t0.y, t1.y), fmaxf(t0.z, t1.z));

    // 三轴区间求交
    tmin = fmaxf(fmaxf(tsm.x, tsm.y), fmaxf(tsm.z, 0.0f)); // 从相机前方（t>=0）开始
    tmax = fminf(fminf(tbg.x, tbg.y), tbg.z);

    return tmax > tmin;
}

// ====== world to uvw in [0,1]======
__device__ float3 worldToUVW(const DeviceVolume &vol, const float3 &pWorld)
{
    //* vol.origin is the min corner of the volume, voxel_size is the size of a voxel, dim is the number of voxels in each direction

#ifndef NDEBUG
    if (vol.dim.x <= 0 || vol.dim.y <= 0 || vol.dim.z <= 0)
    {
        printf("Volume dimension is invalid:(%d, %d, %d)\n",
               vol.dim.x, vol.dim.y, vol.dim.z);
        return make_float3(0.0f, 0.0f, 0.0f);
    }

    const float eps = 1e-6f;
    if (fabs(vol.voxel_size.x) < eps ||
        fabs(vol.voxel_size.y) < eps ||
        fabs(vol.voxel_size.z) < eps)
    {
        printf("Invalid voxel size: (%.6f, %.6f, %.6f)\n",
               vol.voxel_size.x, vol.voxel_size.y, vol.voxel_size.z);
        return make_float3(0.0f, 0.0f, 0.0f);
    }
#endif

    float3 inv_voxel = make_float3(
        1.0f / (vol.voxel_size.x + 1e-6f * signbit(vol.voxel_size.x)),
        1.0f / (vol.voxel_size.y + 1e-6f * signbit(vol.voxel_size.y)),
        1.0f / (vol.voxel_size.z + 1e-6f * signbit(vol.voxel_size.z)));

    float3 idx = make_float3(
        (pWorld.x - vol.origin.x) * inv_voxel.x,
        (pWorld.y - vol.origin.y) * inv_voxel.y,
        (pWorld.z - vol.origin.z) * inv_voxel.z);

    // Normalized to [0,1]
    float3 dimm1 = make_float3(
        __int2float_rn(vol.dim.x - 1),
        __int2float_rn(vol.dim.y - 1),
        __int2float_rn(vol.dim.z - 1));

    float3 uvw = make_float3(
        fminf(fmaxf(idx.x / dimm1.x, 0.0f), 1.0f),
        fminf(fmaxf(idx.y / dimm1.y, 0.0f), 1.0f),
        fminf(fmaxf(idx.z / dimm1.z, 0.0f), 1.0f));

#ifdef DEBUG_UVW
    if (isnan(uvw.x) || isnan(uvw.y) || isnan(uvw.z))
    {
        printf("Invalid UVW: pWorld(%.3f,%.3f,%.3f) -> idx(%.3f,%.3f,%.3f)\n",
               pWorld.x, pWorld.y, pWorld.z, idx.x, idx.y, idx.z);
    }
#endif
    return uvw;
}

__device__ __forceinline__ float sampleField(const DeviceVolume &vol, const float3 uvw)
{
    // volme data is scalar
    // if normalizedCoordinates=0，modify to "tex3D<float>(vol.field_tex, x, y, z)"
    return tex3D<float>(vol.field_tex, uvw.x, uvw.y, uvw.z);
}

__device__ __forceinline__ float3 sampleGradient(const DeviceVolume &vol, const float3 uvw)
{
    float4 f4 = tex3D<float4>(vol.grad_tex, uvw.x, uvw.y, uvw.z);
    return f3(f4.x, f4.y, f4.z);
}

__device__ __forceinline__ float4 sampleTF(const DeviceTF &tf, float value)
{
    // 把 value_range/domain 映射到 [0,1] 采样 1D TF
    float t = (value - tf.domain.x) / (tf.domain.y - tf.domain.x + 1e-8f);
    t = clampf(t, 0.f, 1.f);
    float4 c = tex1D<float4>(tf.tf1D, t);
    return c;
}
__device__ float sampleVolume(const float *volume, int3 dim, float3 pos)
{
    float x = pos.x * (dim.x - 1);
    float y = pos.y * (dim.y - 1);
    float z = pos.z * (dim.z - 1);

    int x0 = floor(x), x1 = min(x0 + 1, dim.x - 1);
    int y0 = floor(y), y1 = min(y0 + 1, dim.y - 1);
    int z0 = floor(z), z1 = min(z0 + 1, dim.z - 1);

    auto at = [&](int xi, int yi, int zi)
    {
        return volume[(zi * dim.y + yi) * dim.x + xi];
    };

    float c000 = at(x0, y0, z0);
    float c100 = at(x1, y0, z0);
    float c010 = at(x0, y1, z0);
    float c110 = at(x1, y1, z0);
    float c001 = at(x0, y0, z1);
    float c101 = at(x1, y0, z1);
    float c011 = at(x0, y1, z1);
    float c111 = at(x1, y1, z1);

    // trilerp(v000, v100, v010, v110, v001, v101, v011, v111, x, y, z);
    float dx = x - x0, dy = y - y0, dz = z - z0;

    float c00 = c000 * (1 - dx) + c100 * dx;
    float c10 = c010 * (1 - dx) + c110 * dx;
    float c0 = c00 * (1 - dy) + c10 * dy;

    float c01 = c001 * (1 - dx) + c101 * dx;
    float c11 = c011 * (1 - dx) + c111 * dx;
    float c1 = c01 * (1 - dy) + c11 * dy;

    c1 = c01 * (1 - dy) + c11 * dy;
    return c0 * (1 - dz) + c1 * dz;
}

__device__ __forceinline__ void compositeFrontToBack(float4 sampledPremulRGBA,
                                                     float opacityScale,
                                                     float4 &accum)
{
    float a_i = clampf(sampledPremulRGBA.w * opacityScale, 0.f, 1.0f);
    float oneMinusAaccum = 1.0f - accum.w;

    // === Lacc = Lacc + (1 - alphaacc)Li
    accum.x += oneMinusAaccum * sampledPremulRGBA.x;
    accum.y += oneMinusAaccum * sampledPremulRGBA.y;
    accum.z += oneMinusAaccum * sampledPremulRGBA.z;
    accum.w += oneMinusAaccum * a_i;
}

//! ===== render kernel ====
__global__ void volumeRendererKernel(const DeviceScene scene,
                                     uchar4 *output,
                                     int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    float3 ro, rd;
    scene.d_camera.generateRay(x, y, width, height, ro, rd);

    float tmin, tmax;
    if (!intersectAABB(ro, rd, scene.clipMin, scene.clipMax, tmin, tmax))
    {
        output[y * width + x] = make_uchar4(0, 0, 0, 255);
        return;
    }

    float4 accum = f4(0, 0, 0, 0);
    float stepWorld = fmaxf(scene.step_size, 1e-4f);
    tmin = fmaxf(tmin, 0.0f);
    const float terminate = 0.98f;
    float maxVal = -1e30f;
    float iso = scene.isoValue;

    // ===  ray march  ===
    for (float t = tmin; t <= tmax; t += stepWorld)
    {
        float3 pWorld = add3(ro, mulS(rd, stepWorld));
        float3 uvw = worldToUVW(scene.d_volume, pWorld);

        if (uvw.x < 0.f || uvw.x > 1.f ||
            uvw.y < 0.f || uvw.y > 1.f ||
            uvw.z < 0.f || uvw.z > 1.f)
            continue;

        float s = sampleField(scene.d_volume, uvw);

        if (scene.mode == 0)
        { // -- mode 0: volume rendering
            float4 c = sampleTF(scene.d_tf, s);

            float a = c.w * scene.d_volume.density_scale;

            float4 permul = f4(c.x * a, c.y * a, c.z * a, a);

            compositeFrontToBack(permul, scene.opacityScale, accum);

            if (accum.w > terminate)
                break;
        }
        else if (scene.mode == 1)
        { // -- mode 1: isosurface rendering
        }
        else if (scene.mode == 2)
        { // -- mode 2: MIP
            maxVal = fmaxf(maxVal, s);
            continue;
        }
    }

    // write output
    float3 color;
    if (scene.mode == 0)
    {
        // Total light intensity divided by total opacity
        color = accum.w > 1e-6f ? f3(accum.x / accum.w, accum.y / accum.w, accum.z / accum.w) : f3(0.f, 0.f, 0.f);
        output[y * width + x] = make_uchar4(
            (unsigned char)clampf(color.x * 255.f, 0.f, 255.f),
            (unsigned char)clampf(color.y * 255.f, 0.f, 255.f),
            (unsigned char)clampf(color.z * 255.f, 0.f, 255.f),
            255);
        return;
    }
}
